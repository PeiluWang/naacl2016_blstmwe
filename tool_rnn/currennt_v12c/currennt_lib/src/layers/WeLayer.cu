
#ifdef _MSC_VER
#   pragma warning (disable: 4244) // thrust/iterator/iterator_adaptor.h(121): warning C4244: '+=' : conversion from '__int64' to 'int', possible loss of data
#endif

#include "WeLayer.hpp"
#include "../Configuration.hpp"

#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/for_each.h>

#include <stdexcept>

#include <boost/random/normal_distribution.hpp>
#include <boost/random/uniform_real_distribution.hpp>
#include <boost/random/mersenne_twister.hpp>

#include <typeinfo>

#include <fstream>


struct saxpy_functor
{
	const real_t a;

	saxpy_functor(real_t _a) : a(_a) {}

	__host__ __device__
		real_t operator()(const real_t& x, const real_t& y) const { 
			return x - a*y;
		}
};


namespace layers {

	template <typename TDevice>
    WeLayer<TDevice>::WeLayer(
        const helpers::JsonValue &layerChild, 
        const helpers::JsonValue &weightsSection,
        Layer<TDevice> &precedingLayer)
        : TrainableLayer<TDevice>(layerChild, weightsSection, 1, 0, precedingLayer)
    {
		
		const Configuration &config = Configuration::instance();
		vocab_size=config.vocabSize();
		if(vocab_size<=0){
			throw std::runtime_error(std::string("vocab_size<=0!"));
		}
		int we_dim=this->size();
		//init we_weights
		Cpu::real_vector weights(vocab_size*we_dim);

		bool trainingmode=config.trainingMode();
		if(trainingmode){// init we weights randomly
			static boost::mt19937 *gen = NULL;
			if (!gen) {
				gen = new boost::mt19937;
				gen->seed(config.randomSeed());
			}
            
			if (config.weightsDistributionType() == Configuration::DISTRIBUTION_UNIFORM) {
				real_t range = config.weightsDistributionUniformMax() - config.weightsDistributionUniformMin();
				boost::random::uniform_real_distribution<real_t> dist(0, range);
				for (size_t i = 0; i < weights.size(); ++i)
					weights[i] = dist(*gen) + config.weightsDistributionUniformMin();
			}
			else {
				boost::random::normal_distribution<real_t> dist(config.weightsDistributionNormalMean(), config.weightsDistributionNormalSigma());
				for (size_t i = 0; i < weights.size(); ++i)
					weights[i] = dist(*gen);
			}
		}else{// load from file
			std::string wedict_file=config.networkFile()+".we";
			std::cout<<"\nload wedict: "<<wedict_file<<std::endl;
			std::ifstream fin(wedict_file);
			std::string line;
			int i=0;
			int word_num=0;
			while(std::getline(fin,line)){
				int s=0;
				int e=0;
				int senlen=line.length();
				while(true){
					e=(int)line.find(" ",s);
					if(e<0){
						break;
					}
					std::string value=line.substr(s,e-s);
					float v=(float)std::atof(value.c_str());
					weights[i]=v;
					i+=1;
					s=e+1;
				}
				word_num+=1;
			}
			std::cout<<"load complete. word num:"<<word_num<<" total value:"<<i<<std::endl;
		}

		//copy weights to we_weights
		we_weights=weights;

		n_learningrate=config.learningRate();
		
    }

    template <typename TDevice>
    WeLayer<TDevice>::~WeLayer()
    {
    }

	template <typename TDevice>
    const std::string& WeLayer<TDevice>::type() const
    {
        static std::string s="welayer";
        return s;
		
    }
	
	template <typename TDevice>
    void WeLayer<TDevice>::loadSequences(const data_sets::DataSetFraction &fraction)
    {
		Layer<TDevice>::loadSequences(fraction);
		thrust::fill(this->_outputs().begin(),this->_outputs().end(),0);

		const Cpu::int_vector& inputs=fraction.inputWords();
		v_inputwords=inputs;
		int parallelSequences=this->parallelSequences();
		int seqnum=fraction.numSequences();
		//int inputdim=fraction.inputPatternSize();
		int we_size=this->size();
		for(int i=0;i<seqnum;++i){
			int seqlength=fraction.seqInfo(i).length;
			for(int timestep=0;timestep<seqlength;++timestep){
				int wordid=inputs[timestep*parallelSequences+i];
				if(wordid==-1){
					throw std::runtime_error(std::string("OOV! in loadSequence"));
					//continue;
				}
				thrust::copy(we_weights.begin()+wordid*we_size,
					we_weights.begin()+(wordid+1)*we_size,
					this->_outputs().begin()+we_size*(timestep*parallelSequences+i));
			}
		}
    }

	template <typename TDevice>
    void WeLayer<TDevice>::computeForwardPass()
    {
	}

	template <typename TDevice>
    void WeLayer<TDevice>::computeBackwardPass()
    {
		int we_size=this->size();
		saxpy_functor fn(n_learningrate);
        // update the we_weights
		int nn=v_inputwords.size();
		int ne=this->outputErrors().size();
		int nw=we_weights.size();
		for(int i=0;i<v_inputwords.size();++i){
			try{
			int wordid=v_inputwords[i];
			if(wordid<0){
				continue;
			}
			thrust::transform(we_weights.begin()+wordid*we_size,
				we_weights.begin()+(wordid+1)*we_size,
				this->outputErrors().begin()+i*we_size,
				we_weights.begin()+wordid*we_size,
				fn);

			}catch(...){
				int wordid=v_inputwords[i];
				int a=wordid;
				throw std::runtime_error(std::string("update we weights error!"));
			}
		}
		
    }

	template <typename TDevice>
	int WeLayer<TDevice>::vocabSize()
	{
		return vocab_size;
	}

	template <typename TDevice>
	typename TDevice::real_vector& WeLayer<TDevice>::weWeights()
	{
		return we_weights;
	}

	 template class WeLayer<Cpu>;
	 template class WeLayer<Gpu>;
}