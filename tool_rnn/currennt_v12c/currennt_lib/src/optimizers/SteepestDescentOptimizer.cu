/******************************************************************************
 * Copyright (c) 2013 Johannes Bergmann, Felix Weninger, Bjoern Schuller
 * Institute for Human-Machine Communication
 * Technische Universitaet Muenchen (TUM)
 * D-80290 Munich, Germany
 *
 * This file is part of CURRENNT.
 *
 * CURRENNT is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * CURRENNT is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with CURRENNT.  If not, see <http://www.gnu.org/licenses/>.
 *****************************************************************************/

#ifdef _MSC_VER
#   pragma warning (disable: 4244) // thrust/iterator/iterator_adaptor.h(121): warning C4244: '+=' : conversion from '__int64' to 'int', possible loss of data
#endif

#include "SteepestDescentOptimizer.hpp"
#include "../layers/TrainableLayer.hpp"
#include "../helpers/getRawPointer.cuh"
#include "../rapidjson/document.h"

#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>


namespace internal {
namespace {

    struct UpdateWeightFn
    {
        real_t learningRate;
        real_t momentum;

        const real_t *weights;
        const real_t *weightUpdates;
        real_t       *weightDeltas;

        __host__ __device__ real_t operator() (const int &weightIdx)
        {
            // calculate and store the weight delta
            real_t delta = momentum * weightDeltas[weightIdx] - (1-momentum)* learningRate * weightUpdates[weightIdx];
            weightDeltas[weightIdx] = delta;

            // calculate the new weight
            real_t newWeight = weights[weightIdx] + delta;

            return newWeight;
        }
    };

} // anonymous namespace
} // namespace internal


namespace optimizers {

    template <typename TDevice>
    void SteepestDescentOptimizer<TDevice>::_updateWeights()
    {
        internal::UpdateWeightFn updateWeightFn;
        updateWeightFn.learningRate = m_learningRate;
        updateWeightFn.momentum     = m_momentum;

        for (size_t i = 1; i < this->_neuralNetwork().layers().size()-1; ++i) {
        	layers::TrainableLayer<TDevice> *layer = dynamic_cast<layers::TrainableLayer<TDevice>*>(this->_neuralNetwork().layers()[i].get());
			if(layer->type()=="welayer"){
				continue;
			}
            updateWeightFn.weights       = helpers::getRawPointer(layer->weights());
            updateWeightFn.weightUpdates = helpers::getRawPointer(this->_curWeightUpdates()[i]);
            updateWeightFn.weightDeltas  = helpers::getRawPointer(m_weightDeltas[i]);

            thrust::transform(
                thrust::counting_iterator<int>(0),
                thrust::counting_iterator<int>((int)layer->weights().size()),
                layer->weights().begin(),
                updateWeightFn
                );
        }
    }

    template <typename TDevice>
    SteepestDescentOptimizer<TDevice>::SteepestDescentOptimizer(
        NeuralNetwork<TDevice> &neuralNetwork, data_sets::DataSet &trainingSet, data_sets::DataSet &validationSet,
        data_sets::DataSet &testSet, int maxEpochs, int maxEpochsNoBest, int validateEvery, int testEvery, 
        real_t learningRate, real_t momentum)
        : Optimizer<TDevice>(neuralNetwork, trainingSet, validationSet, testSet, maxEpochs, maxEpochsNoBest, validateEvery, testEvery)
        , m_learningRate    (learningRate)
        , m_momentum        (momentum)
    {
        // intialize the weight deltas vectors with zeros
        m_weightDeltas = this->_curWeightUpdates();
        for (size_t i = 0; i < m_weightDeltas.size(); ++i)
            thrust::fill(m_weightDeltas[i].begin(), m_weightDeltas[i].end(), 0);
    }

    template <typename TDevice>
    SteepestDescentOptimizer<TDevice>::~SteepestDescentOptimizer()
    {
    }

    template <typename TDevice>
    void SteepestDescentOptimizer<TDevice>::exportState(const helpers::JsonDocument &jsonDoc) const
    {
        Optimizer<TDevice>::exportState(jsonDoc);

        Optimizer<TDevice>::_exportWeights(jsonDoc, "steepest_descent_optimizer_weight_deltas", m_weightDeltas);
    }

    template <typename TDevice>
    void SteepestDescentOptimizer<TDevice>::importState(const helpers::JsonDocument &jsonDoc)
    {
        Optimizer<TDevice>::importState(jsonDoc);

        Optimizer<TDevice>::_importWeights(jsonDoc, "steepest_descent_optimizer_weight_deltas", &m_weightDeltas);
    }


    // explicit template instantiations
    template class SteepestDescentOptimizer<Cpu>;
    template class SteepestDescentOptimizer<Gpu>;

} // namespace optimizers
