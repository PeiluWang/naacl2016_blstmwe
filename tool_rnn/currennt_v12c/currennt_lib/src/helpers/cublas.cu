/******************************************************************************
 * Copyright (c) 2013 Johannes Bergmann, Felix Weninger, Bjoern Schuller
 * Institute for Human-Machine Communication
 * Technische Universitaet Muenchen (TUM)
 * D-80290 Munich, Germany
 *
 * This file is part of CURRENNT.
 *
 * CURRENNT is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * CURRENNT is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with CURRENNT.  If not, see <http://www.gnu.org/licenses/>.
 *****************************************************************************/

#include "cublas.hpp"

#include <hipblas.h>

#include <stdexcept>


namespace internal {
namespace {

    hipblasHandle_t getCublasHandle()
    {
        static hipblasHandle_t handle = 0;
        if (!handle) {
            hipblasStatus_t res = hipblasCreate(&handle);
            if (res != HIPBLAS_STATUS_SUCCESS)
                throw std::runtime_error("Could not create CUBLAS handle");
        }

        return handle;
    }

} // anonymous namespace
} // namespace internal


namespace helpers {
namespace cublas {

    template <>
    void multiplyMatrices<float>(
        bool transposeA, bool transposeB,
        int m, int n, int k,
        const float *matrixA, int ldA,
        const float *matrixB, int ldB,
        float *matrixC, int ldC,
        bool addOldMatrixC
        )
    {
        float alpha = 1;
        float beta  = (addOldMatrixC ? 1.0f : 0.0f);

        hipblasStatus_t res = hipblasSgemm(
            /* handle */ internal::getCublasHandle(),
            /* transa */ transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
            /* transb */ transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
            /* m      */ m,
            /* n      */ n,
            /* k      */ k,
            /* alpha  */ &alpha,
            /* A      */ matrixA,
            /* lda    */ ldA,
            /* B      */ matrixB,
            /* ldb    */ ldB,
            /* beta   */ &beta,
            /* C      */ matrixC,
            /* ldc    */ ldC
            );

        if (res != HIPBLAS_STATUS_SUCCESS)
            throw std::runtime_error("CUBLAS matrix multiplication failed");
    }

    template <>
    void multiplyMatrices<double>(
        bool transposeA, bool transposeB,
        int m, int n, int k,
        const double *matrixA, int ldA,
        const double *matrixB, int ldB,
        double *matrixC, int ldC,
        bool addOldMatrixC
        )
    {
        double alpha = 1;
        double beta  = (addOldMatrixC ? 1 : 0);

        hipblasStatus_t res = hipblasDgemm(
            /* handle */ internal::getCublasHandle(),
            /* transa */ transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
            /* transb */ transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
            /* m      */ m,
            /* n      */ n,
            /* k      */ k,
            /* alpha  */ &alpha,
            /* A      */ matrixA,
            /* lda    */ ldA,
            /* B      */ matrixB,
            /* ldb    */ ldB,
            /* beta   */ &beta,
            /* C      */ matrixC,
            /* ldc    */ ldC
            );

        if (res != HIPBLAS_STATUS_SUCCESS)
            throw std::runtime_error("CUBLAS matrix multiplication failed");
    }

} // namespace cublas
} // namespace helpers
