#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2013 Johannes Bergmann, Felix Weninger, Bjoern Schuller
 * Institute for Human-Machine Communication
 * Technische Universitaet Muenchen (TUM)
 * D-80290 Munich, Germany
 *
 * This file is part of CURRENNT.
 *
 * CURRENNT is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * CURRENNT is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with CURRENNT.  If not, see <http://www.gnu.org/licenses/>.
 *****************************************************************************/

#include "InputLayer.hpp"
#include "../Configuration.hpp"
#include "../helpers/Matrix.hpp"
#include "../helpers/NumericLimits.cuh"
#include "../helpers/getRawPointer.cuh"
#include "../activation_functions/Tanh.cuh"
#include "../activation_functions/Logistic.cuh"

#include <boost/random/normal_distribution.hpp>
#include <boost/random/uniform_real_distribution.hpp>
#include <boost/random/mersenne_twister.hpp>

#include <thrust/iterator/counting_iterator.h>
#include <boost/lexical_cast.hpp>
#include <stdexcept>
#include <math.h>
#include <fstream>

struct UpdateWeWeights
{
	real_t learningrate;
	real_t momentum;

	real_t* weweights;
	real_t* weupdateweights;
	real_t* lastweupdateweights;

	__host__ __device__
	real_t operator()(const int &id) const { 
		lastweupdateweights[id]=momentum*lastweupdateweights[id]-(1-momentum)*learningrate*weupdateweights[id];

		real_t newweweights=weweights[id]+lastweupdateweights[id];
		return newweweights;
	}
};

struct UpdateFeatWeights
{
	real_t learningrate;
	real_t momentum;

	real_t* featweights;
	real_t* featupdateweights;
	real_t* lastfeatupdateweights;

	__host__ __device__
	real_t operator()(const int &id) const { 
		lastfeatupdateweights[id]=momentum*lastfeatupdateweights[id]-(1-momentum)*learningrate*featupdateweights[id];

		real_t newweweights=featweights[id]+lastfeatupdateweights[id];
		return newweweights;
	}
};

struct LogisticForward
{
	__host__ __device__
	real_t operator()(const real_t& x) const { 
		real_t n_x=activation_functions::Tanh::fn(x);
		//real_t n_x=log(1.0+exp(x));
		/*
		real_t n_x=x;
		if(x<0){
			n_x=0;
		}*/
		return n_x;
	}
};

struct ComputeDelta
    {
        // since calculating the derivatives is very cheap for our activation functions, 
        // we simple calculate the deltas of all timesteps, including dummies
        
        __host__ __device__ void operator() (const thrust::tuple<real_t&, const real_t&> &t) const
        {
            real_t delta = activation_functions::Tanh::deriv(t.get<1>()) * t.get<0>();
			//real_t delta = 1.0/(1.0+exp(-t.get<1>()))*t.get<0>(); //rectifier
			/*
			real_t delta=t.get<0>();
			if(t.get<1>()<0){
				delta=0;
			}*/
            t.get<0>() = delta;
        }
    };

namespace layers {

    template <typename TDevice>
    InputLayer<TDevice>::InputLayer(const helpers::JsonValue &layerChild, int parallelSequences, int maxSeqLength)
        : Layer<TDevice>(layerChild, parallelSequences, maxSeqLength)
    {
		const Configuration &config = Configuration::instance();
		int vocab_size=config.vocabSize();
		inputWeDim=config.inputWeDim();
		inputFeatDim=config.inputFeatDim();

		assert(vocab_size>0);
		assert(inputWeDim>0);
		assert(inputFeatDim>0);

		// init we_weights
		Cpu::real_vector weights(vocab_size*inputWeDim);
		Cpu::real_vector fweights(inputFeatDim*inputWeDim);

		bool loadweweights=false;
		if(config.loadWeweightsFile()!="none"){
			loadweweights=true;
		}
		if(!loadweweights){// init we weights randomly
			std::cout<<"\ninit weweights randomly\n"<<std::endl;
			static boost::mt19937 *gen = NULL;
			if (!gen) {
				gen = new boost::mt19937;
				gen->seed(config.randomSeed());
			}
			if (config.weightsDistributionType() == Configuration::DISTRIBUTION_UNIFORM) {
				real_t range = config.weightsDistributionUniformMax() - config.weightsDistributionUniformMin();
				boost::random::uniform_real_distribution<real_t> dist(0, range);
				for (size_t i = 0; i < weights.size(); ++i)
					weights[i] = dist(*gen) + config.weightsDistributionUniformMin();
			}
			else {
				boost::random::normal_distribution<real_t> dist(config.weightsDistributionNormalMean(), config.weightsDistributionNormalSigma());
				for (size_t i = 0; i < weights.size(); ++i)
					weights[i] = dist(*gen);
			}
		}else{// load we weights from file
			std::string wedict_file=config.loadWeweightsFile();
			std::cout<<"\nload weweights: "<<wedict_file<<std::endl;
			std::ifstream fin(wedict_file);
			if(!fin){
				std::cerr<<"load wedict_file exception: "<<wedict_file<<std::endl;
				throw std::runtime_error(std::string("wedict_file not exist!"));
			}
			std::string line;
			int i=0;
			int word_num=0;
			while(std::getline(fin,line)){
				int s=0;
				int e=0;
				int senlen=line.length();
				while(true){
					e=(int)line.find(" ",s);
					if(e<0){
						break;
					}
					std::string value=line.substr(s,e-s);
					float v=(float)std::atof(value.c_str());
					weights[i]=v;
					i+=1;
					s=e+1;
				}
				word_num+=1;
			}
			std::cout<<"load complete. word num:"<<word_num<<" total value:"<<i<<std::endl;
		}
		//copy weights to we_weights
		we_weights=weights;
		last_weweightUpdates=weights;

		bool loadfeatweights=false;
		if(config.loadFeatweightsFile()!="none"){
			loadfeatweights=true;
		}
		if(!loadfeatweights){// init feat weights randomly
			std::cout<<"\ninit featweights randomly\n"<<std::endl;
			static boost::mt19937 *gen = NULL;
			if (!gen) {
				gen = new boost::mt19937;
				gen->seed(config.randomSeed());
			}
			if (config.weightsDistributionType() == Configuration::DISTRIBUTION_UNIFORM) {
				real_t range = config.weightsDistributionUniformMax() - config.weightsDistributionUniformMin();
				boost::random::uniform_real_distribution<real_t> dist(0, range);
				for (size_t i = 0; i < fweights.size(); ++i)
					fweights[i] = dist(*gen) + config.weightsDistributionUniformMin();
			}
			else {
				boost::random::normal_distribution<real_t> dist(config.weightsDistributionNormalMean(), config.weightsDistributionNormalSigma());
				for (size_t i = 0; i < fweights.size(); ++i)
					fweights[i] = dist(*gen);
			}
		}else{// load feat weights from file
			std::string featweights_file=config.loadFeatweightsFile();
			std::cout<<"\nload featweights: "<<featweights_file<<std::endl;
			std::ifstream fin(featweights_file);
			if(!fin){
				std::cerr<<"load featweights exception: "<<featweights_file<<std::endl;
				throw std::runtime_error(std::string("featweights not exist!"));
			}
			std::stringstream buffer;
			buffer << fin.rdbuf();
			std::string line(buffer.str());

			int i=0;
			int s=0;
			int e=0;
			while(true){
				e=(int)line.find(" ",s);
				if(e<0){
					break;
				}
				std::string value=line.substr(s,e-s);
				float v=(float)std::atof(value.c_str());
				fweights[i]=v;
				i+=1;
				s=e+1;
			}
			
			std::cout<<"load complete. weights size:"<<i<<std::endl;
		}
		//copy weights to feat_weights
		feat_weights=fweights;
		feat_weightUpdates=fweights;

		last_featweightUpdates=fweights;
		thrust::fill(last_weweightUpdates.begin(), last_weweightUpdates.end(), 0);
		thrust::fill(last_featweightUpdates.begin(), last_featweightUpdates.end(), 0);

		v_outputErrors = Cpu::real_vector(this->_outputs().size(), (real_t)0);
    }

    template <typename TDevice>
    InputLayer<TDevice>::~InputLayer()
    {
    }

    template <typename TDevice>
    const std::string& InputLayer<TDevice>::type() const
    {
        static const std::string s("input");
        return s;
    }

    template <typename TDevice>
    void InputLayer<TDevice>::loadSequences(const data_sets::DataSetFraction &fraction)
    {
        Layer<TDevice>::loadSequences(fraction);
		thrust::fill(this->_outputs().begin(),this->_outputs().end(),0);

		int parallelnum=this->parallelSequences();
		int seqnum=fraction.numSequences();
		int layersize=this->size();

		assert(layersize==inputWeDim);

		v_inputwords=fraction.inputWords();
		v_inputfeats=fraction.inputFeats();

		// load word embedding
		for(int pid = 0; pid < seqnum; ++pid){
			int seqlength=fraction.seqInfo(pid).length;
			for(int stepid = 0; stepid < seqlength; ++stepid){
				int t=stepid * parallelnum + pid;
				//load input word, applying word embedding
				int wordid=v_inputwords[t];
				assert(wordid>=0);
				thrust::copy_n(we_weights.begin()+wordid*inputWeDim,
					inputWeDim,
					this->_outputs().begin()+inputWeDim * t);
			}
		}
    }

    template <typename TDevice>
    void InputLayer<TDevice>::computeForwardPass()
    {
		{{
			// calculate inputFeat forward
			helpers::Matrix<TDevice> weightsMatrix (&feat_weights, inputFeatDim, inputWeDim);
			helpers::Matrix<TDevice> plOutputsMatrix(&v_inputfeats, inputFeatDim, this->curMaxSeqLength() * this->parallelSequences());
			helpers::Matrix<TDevice> outputsMatrix  (&this->_outputs(), inputWeDim, this->curMaxSeqLength() * this->parallelSequences());

			outputsMatrix.addProduct(weightsMatrix, true, plOutputsMatrix, false);
		}}
		/*
		{{
			// calculate logistic
			LogisticForward fn;
			thrust::transform(this->_outputs().begin(),
				this->_outputs().end(),
				this->_outputs().begin(),
				fn);
		}}
		*/
    }

    template <typename TDevice>
    void InputLayer<TDevice>::computeBackwardPass()
    {
		float n_learningrate=Configuration::instance().learningRate();
		int layersize = this->size();
		float momentum=Configuration::instance().momentum();
		/*
		// compute delta
		{{
			ComputeDelta fndelta;

            int n = this->curMaxSeqLength() * this->parallelSequences() * this->size();

            thrust::for_each(
                thrust::make_zip_iterator(thrust::make_tuple(this->outputErrors().begin(),   this->outputs().begin())),
                thrust::make_zip_iterator(thrust::make_tuple(this->outputErrors().begin()+n, this->outputs().begin()+n)),
                fndelta
                );
		}}
		*/
		UpdateWeWeights fn;
		fn.learningrate=n_learningrate;
		fn.momentum=momentum;
		real_t* weweights=helpers::getRawPointer(we_weights);
		real_t* outputErrors=helpers::getRawPointer(v_outputErrors);
		real_t* lastweupdateweights=helpers::getRawPointer(last_weweightUpdates);

        // update we_weights
		for(int i=0;i<v_inputwords.size();++i){
			try{
			int wordid=v_inputwords[i];
			if(wordid<0){
				continue;
			}
			int offset=wordid*inputWeDim;
			fn.weweights=weweights+offset;
			fn.weupdateweights=outputErrors+i*layersize;
			fn.lastweupdateweights=lastweupdateweights+offset;

			thrust::transform(
				thrust::counting_iterator<int>(0),
				thrust::counting_iterator<int>(inputWeDim),
				we_weights.begin()+offset,
				fn);
			}catch(...){
				int wordid=v_inputwords[i];
				std::cerr<<"update inputLayer we weights error! wordid:"<<wordid<<" i:"<<i<<std::endl;
				throw std::runtime_error(std::string("update we weights error!"));
			}
		}
		// compute the weight updates
        {{
            helpers::Matrix<TDevice> weightUpdatesMatrix(&feat_weightUpdates, inputFeatDim, inputWeDim);
            helpers::Matrix<TDevice> plOutputsMatrix (&v_inputfeats, inputFeatDim, this->curMaxSeqLength() * this->parallelSequences());
            helpers::Matrix<TDevice> deltasMatrix (&this->outputErrors(), inputWeDim, this->curMaxSeqLength() * this->parallelSequences());

            weightUpdatesMatrix.assignProduct(plOutputsMatrix, false, deltasMatrix, true);
        }}
		// update feat_weights
		{{
			UpdateFeatWeights fn2;
			fn2.learningrate=n_learningrate;
			fn2.momentum=momentum;
			fn2.featweights=helpers::getRawPointer(feat_weights);
			fn2.featupdateweights=helpers::getRawPointer(feat_weightUpdates);
			fn2.lastfeatupdateweights=helpers::getRawPointer(last_featweightUpdates);
			
			thrust::transform(
				thrust::counting_iterator<int>(0),
				thrust::counting_iterator<int>((int)feat_weights.size()),
				feat_weights.begin(),
				fn2);
		}}
    }

	template <typename TDevice>
    typename InputLayer<TDevice>::real_vector& InputLayer<TDevice>::outputErrors()
    {
        return v_outputErrors;
    }

	template <typename TDevice>
    typename InputLayer<TDevice>::real_vector& InputLayer<TDevice>::weWeights()
    {
        return we_weights;
    }
	
	template <typename TDevice>
    typename InputLayer<TDevice>::real_vector& InputLayer<TDevice>::featWeights()
    {
        return feat_weights;
    }

    // explicit template instantiations
    template class InputLayer<Cpu>;
    template class InputLayer<Gpu>;

} // namespace layers
